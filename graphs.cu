#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_ERROR_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__ << " - " \
                      << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    }

__global__ void kernel_A(int *data) 
{
    printf("hellur A %d\n",*data);
}
__global__ void kernel_B(int *data) 
{
    printf("hellur B %d\n", *data);
}
__global__ void kernel_C() {}
__global__ void kernel_D() {}
__global__ void kernel_E() {}

int main()
{
    hipGraph_t graph;
    hipGraphExec_t graph_exec;
    hipGraphNode_t nodeA, nodeB;
    hipKernelNodeParams k_params = {0}; 

    int a = 1, b = 2;
    int *d_a_ptr, *d_b_ptr; 
    hipMalloc(&d_a_ptr, sizeof(int));
    hipMalloc(&d_b_ptr, sizeof(int));
    hipMemcpy(d_a_ptr, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b_ptr, &b, sizeof(int), hipMemcpyHostToDevice);


    void **kernel_args = (void **)malloc(sizeof(void*));
    
    k_params.func = kernel_A; 
    k_params.gridDim = dim3 (1,1,1);
    k_params.blockDim = dim3(1,1,1);
    k_params.sharedMemBytes = 0;
    k_params.kernelParams = kernel_args;
    kernel_args[0] = &d_a_ptr;
    k_params.extra = nullptr; // No extra parameters

    hipGraphCreate(&graph, 0);
    
    hipGraphAddKernelNode(&nodeA, graph, NULL, 0, &k_params);
    
    k_params.func = kernel_B; 
    kernel_args[0] = &d_b_ptr;
    hipGraphAddKernelNode(&nodeB, graph, NULL, 0, &k_params);
    
    
    hipGraphInstantiate(&graph_exec, graph);
    hipGraphLaunch(graph_exec,0);
    
    hipFree(d_a_ptr);
    hipFree(d_b_ptr);
    hipGraphDestroy(graph);
    hipGraphExecDestroy(graph_exec);

}