#include "hip/hip_runtime.h"
__global__ void primary_kernel() {
   // Initial work that should finish before starting secondary kernel

   // Trigger the secondary kernel
   cudaTriggerProgrammaticLaunchCompletion();

   // Work that can coincide with the secondary kernel
}

__global__ void secondary_kernel()
{
   // Independent work

   // Will block until all primary kernels the secondary kernel is dependent on have completed and flushed results to global memory
   cudaGridDependencySynchronize();

   // Dependent work
}

int main()
{
hipLaunchAttribute attribute[1];
attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
attribute[0].val.programmaticStreamSerializationAllowed = 1;
configSecondary.attrs = attribute;
configSecondary.numAttrs = 1;

primary_kernel<<<grid_dim, block_dim, 0, stream>>>();
cudaLaunchKernelEx(&configSecondary, secondary_kernel);
}