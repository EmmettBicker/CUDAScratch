#include <hip/hip_runtime.h>
#include <iostream>

__global__ void kernel() {
    printf("Hello from kernel\n");
}

void checkCudaErrors(hipError_t error) {
    if (error != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(error) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    hipStream_t stream1, stream2;
    hipGraph_t graph;

    // Create streams
    checkCudaErrors(hipStreamCreate(&stream1));
    checkCudaErrors(hipStreamCreate(&stream2));

    // Begin capture in stream1
    checkCudaErrors(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));

    // Launch kernel in stream1
    kernel<<<1, 1, 0, stream1>>>();
    checkCudaErrors(hipGetLastError());

    // Record an event in stream1
    hipEvent_t event;
    checkCudaErrors(hipEventCreate(&event));
    checkCudaErrors(hipEventRecord(event, stream1));

    // Wait for the event in stream2
    checkCudaErrors(hipStreamWaitEvent(stream2, event, 0));

   

    // End capture in stream1
    checkCudaErrors(hipStreamEndCapture(stream1, &graph));
 // Launch kernel in stream2
    kernel<<<1, 1, 0, stream2>>>();
    checkCudaErrors(hipGetLastError());
    // Synchronize streams to ensure all operations are completed
    checkCudaErrors(hipStreamSynchronize(stream1));
    checkCudaErrors(hipStreamSynchronize(stream2));

    // Cleanup
    checkCudaErrors(hipStreamDestroy(stream1));
    checkCudaErrors(hipStreamDestroy(stream2));
    checkCudaErrors(hipEventDestroy(event));

    return 0;
}
