#include <hip/hip_runtime.h>
#include <iostream>
#include <limits.h> // For INT_MAX

#define CUDA_CHECK(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __FILE__ << ":" << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

__global__ void k1(int *x)
{
    int b = 4/0;
    for (int i = 0; i < INT_MAX+1; i++) {*x += i % 2 == 0 ? 1 : 0;}
    printf("Hello from %d\n",*x);
}

__global__ void k2(int *x)
{
    for (int i = 0; i < INT_MAX; i++) {*x += i % 2 == 0 ? 1 : 0;}
    printf("Hello from %d\n",*x);
}

void CUDART_CB cb(void* userData)
{
    printf("Callback\n");
}

int main()
{
    hipStream_t st_high, st_low;
    hipStreamCreateWithPriority(&st_high, hipStreamNonBlocking, -5);
    hipStreamCreateWithPriority(&st_low, hipStreamNonBlocking, 1);

    int one_val = 1, two_val = 2, three_val = 3, four_val = 4;
    int *one, *two, *three, *four;
    
    hipMalloc(&one, sizeof(int));
    hipMalloc(&two, sizeof(int));
    hipMalloc(&three, sizeof(int));
    hipMalloc(&four, sizeof(int));

    hipMemcpy(one, &one_val, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(two, &two_val, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(three, &three_val, sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy(four, &four_val, sizeof(int), hipMemcpyHostToDevice);

    k1<<<1,1,0,st_low>>>(three);
    hipError_t error = hipGetLastError();
if (error != hipSuccess) {
    std::cerr << "Kernel launch error: " << hipGetErrorString(error) << std::endl;
    exit(EXIT_FAILURE);
}
    k2<<<1,1,0,st_low>>>(four);
    error = hipGetLastError();
if (error != hipSuccess) {
    std::cerr << "Kernel launch error: " << hipGetErrorString(error) << std::endl;
    exit(EXIT_FAILURE);
}
    hipLaunchHostFunc(st_low, cb, nullptr);


    hipDeviceSynchronize();

    k1<<<1,1,0,st_high>>>(one);
    error = hipGetLastError();
if (error != hipSuccess) {
    std::cerr << "Kernel launch error: " << hipGetErrorString(error) << std::endl;
    exit(EXIT_FAILURE);
}
    k2<<<1,1,0,st_high>>>(two);
    error = hipGetLastError();
if (error != hipSuccess) {
    std::cerr << "Kernel launch error: " << hipGetErrorString(error) << std::endl;
    exit(EXIT_FAILURE);
}
    hipLaunchHostFunc(st_high, cb, nullptr);
    hipDeviceSynchronize();
    


}