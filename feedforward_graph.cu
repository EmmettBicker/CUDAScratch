#include <hip/hip_runtime.h>
#include <>

#include <iostream>

#include "mat_mult.cuh"

int main() {
    int w1 = 2, h1 = 3, w2 = 2, h2 = 2;

    const int num_streams = 2;

    hipStream_t streams[num_streams];
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float arr1[] = {1, 2, 3, 4, 5, 6};
    float arr2[] = {1, 2, 3, 4};

    float *d_mat1, *d_mat2, *d_out;
    size_t sz_mat1 = sizeof(float) * w1 * h1;
    size_t sz_mat2 = sizeof(float) * w2 * h2;
    size_t sz_out = sizeof(float) * h1 * w2;

    hipMalloc((void**)&d_mat1, sz_mat1);
    hipMalloc((void**)&d_mat2, sz_mat2);
    hipMalloc((void**)&d_out, sz_out);

    float* h_out;

    hipHostMalloc((void**)&h_out, sz_out);

    hipMemset(d_out, 0, sz_out);

    // hipEventRecord(start);
    // hipEventRecord(stop);
    // hipEventSynchronize(stop);
    // Use hipMemcpyAsync with specified stream
    hipMemcpyAsync(d_mat1, arr1, sz_mat1, hipMemcpyHostToDevice);
    hipMemcpyAsync(d_mat2, arr2, sz_mat2, hipMemcpyHostToDevice);

    // float ms = 0;
    // hipEventElapsedTime(&ms, start, stop);
    // printf("%f ms\n", ms);

    hipGraph_t graph;
    hipGraphExec_t graph_exec;
    hipGraphNode_t node_A;
    hipGraphCreate(&graph, 0);

    hipKernelNodeParams k_params;
    dim3 gridDim(1, 1);
    dim3 blockDim(h1, w2);

    k_params.func = (void*)mat_mult;
    k_params.gridDim = gridDim;
    k_params.blockDim = blockDim;
    void* k_args[] = {&w1, &h1, &w2, &h2, &d_mat1, &d_mat2, &d_out};
    k_params.kernelParams = k_args;
    k_params.sharedMemBytes = 0;

    hipGraphAddKernelNode(&node_A, graph, NULL, 0, &k_params);

    hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0);

    hipDeviceSynchronize();  // Ensure all previous operations are done
    hipGraphLaunch(graph_exec, 0);
    // mat_mult<<<1,dim3(h1, w2)>>>(w1, h1, w2, h2, d_mat1, d_mat2, d_out);

    hipDeviceSynchronize();

    hipMemcpy(h_out, d_out, sz_out, hipMemcpyDeviceToHost);

    printf("\n");
    for (int i = 0; i < h1 * w2; i++) {
        printf("%d: %f\n", i, h_out[i]);
    }

    for (int i = 0; i < num_streams; i++)
        hipStreamDestroy(streams[i]);
    free(h_out);
    hipFree(d_mat1);
    hipFree(d_mat2);
    hipFree(d_out);
}