#include <iostream>
#include <hip/hip_runtime.h>

__global__ void myKernel(volatile int* dummy)
{
    long long sum = 0;
    for (long long i = 0; i < 100000001; i++)
    {

        sum += i % 2 == 0 ? i : 0;
    }
    *dummy = sum;  // Prevent optimization
}

int main()
{
    hipStream_t stream, stream2;
    hipStreamCreate(&stream);
    hipStreamCreate(&stream2);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int *d_dummy;
    hipMalloc(&d_dummy, sizeof(int));

    myKernel<<<1,1>>>(d_dummy);
    hipDeviceSynchronize();
    hipEventRecord(start);
    
    myKernel<<<1,1,0,stream>>>(d_dummy);
    myKernel<<<1,1, 0, stream2>>>(d_dummy);

   
    hipEventRecord(stop);
    hipEventSynchronize(stop);  // Wait for stop event to complete

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Elapsed time: %f ms\n", ms);

    hipFree(d_dummy);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipStreamDestroy(stream);

    return 0;
}